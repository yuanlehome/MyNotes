#include "hip/hip_runtime.h"
#include "all.h"
#include "common.h"
#include "kernel_utils.cu.h"

// Naive solution as baseline
__global__ void matrixMultiplyKernel_V1(const DATA_TYPE* __restrict__ A,
                                        const DATA_TYPE* __restrict__ B,
                                        DATA_TYPE* __restrict__ C,
                                        int M,
                                        int N,
                                        int K,
                                        DATA_TYPE alpha,
                                        DATA_TYPE beta) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  if (col < N && row < M) {
    DATA_TYPE sum = 0.0;
    for (int k = 0; k < K; k++) {
      sum += A[k + row * K] * B[col + k * N];
    }
    C[col + row * N] = alpha * sum + beta * C[col + row * N];
  }
}

// Use shared memory and avoid bank conflicts
template <int kTileDim>
__global__ void matrixMultiplyKernel_V2(const DATA_TYPE* __restrict__ A,
                                        const DATA_TYPE* __restrict__ B,
                                        DATA_TYPE* __restrict__ C,
                                        int M,
                                        int N,
                                        int K,
                                        DATA_TYPE alpha,
                                        DATA_TYPE beta) {
  __shared__ DATA_TYPE s_a[kTileDim][kTileDim];
  __shared__ DATA_TYPE s_b[kTileDim][kTileDim + 1];
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  if (col < N && row < M) {
    DATA_TYPE sum = 0.0;
    for (int k = 0; k < K; k += kTileDim) {
      s_a[threadIdx.y][threadIdx.x] = A[k + threadIdx.x + row * K];
      s_b[threadIdx.y][threadIdx.x] = B[col + (k + threadIdx.y) * N];
      __syncthreads();
#pragma unroll
      for (int i = 0; i < kTileDim; i++) {
        sum += s_a[threadIdx.y][i] * s_b[i][threadIdx.x];
      }
      __syncthreads();
    }
    C[col + row * N] = alpha * sum + beta * C[col + row * N];
  }
}

// Use vector 4 access
template <int kTileDim>
__global__ void matrixMultiplyKernel_V3(const DATA_TYPE* __restrict__ A,
                                        const DATA_TYPE* __restrict__ B,
                                        DATA_TYPE* __restrict__ C,
                                        int M,
                                        int N,
                                        int K,
                                        DATA_TYPE alpha,
                                        DATA_TYPE beta) {
  __shared__ DATA_TYPE s_a[kTileDim][kTileDim + 1];
  __shared__ DATA_TYPE s_b[kTileDim][kTileDim + 1];
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  if (col < N && row < M) {
    DATA_TYPE sum = 0.0;
    for (int k = 0; k < K; k += kTileDim) {
      s_a[threadIdx.y][threadIdx.x] = A[k + threadIdx.x + row * K];
      s_b[threadIdx.y][threadIdx.x] = B[col + (k + threadIdx.y) * N];
      __syncthreads();
#pragma unroll
      for (int i = 0; i < kTileDim; i++) {
        sum += s_a[threadIdx.y][i] * s_b[i][threadIdx.x];
      }
      __syncthreads();
    }
    C[col + row * N] = alpha * sum + beta * C[col + row * N];
  }
}

void gemm_naive() {
  constexpr uint32_t M = 1024;
  constexpr uint32_t N = 1024;
  constexpr uint32_t K = 512;

  constexpr uint32_t A_SIZE = sizeof(DATA_TYPE) * M * K;
  constexpr uint32_t B_SIZE = sizeof(DATA_TYPE) * K * N;
  constexpr uint32_t C_SIZE = sizeof(DATA_TYPE) * M * N;

  CpuMallocWrapper cpu_allocator;
  DATA_TYPE* h_a = (DATA_TYPE*)cpu_allocator.allocate(A_SIZE);
  DATA_TYPE* h_b = (DATA_TYPE*)cpu_allocator.allocate(B_SIZE);
  DATA_TYPE* h_c = (DATA_TYPE*)cpu_allocator.allocate(C_SIZE);
  std::fill_n(h_a, M * K, 0.5);
  std::fill_n(h_b, K * N, 0.3);
  std::fill_n(h_c, M * N, 0.0);

  GpuMallocWrapper gpu_allocator;
  DATA_TYPE* d_a = (DATA_TYPE*)gpu_allocator.allocate(A_SIZE);
  DATA_TYPE* d_b = (DATA_TYPE*)gpu_allocator.allocate(B_SIZE);
  DATA_TYPE* d_c = (DATA_TYPE*)gpu_allocator.allocate(C_SIZE);
  CUDA_CHECK(hipMemcpy(d_a, h_a, A_SIZE, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_b, h_b, B_SIZE, hipMemcpyHostToDevice));
  utils::fill_n(d_c, M * N, 0.0);

  // CPU results
  DATA_TYPE* real_c = (DATA_TYPE*)cpu_allocator.allocate(C_SIZE);
  std::fill_n(real_c, M * N, 0.0);
  utils::matrixMultiply(h_a, h_b, real_c, M, N, K);

  // GPU results
  utils::performance<GpuTimer>(
      "matrixMultiplyKernel_V1_16x16x16",
      repeats,
      [&] { utils::fill_n(d_c, M * N, 0.0); },
      [&] {
        constexpr int kTileDim = 16;
        dim3 block(kTileDim, kTileDim);
        dim3 grid((M + kTileDim - 1) / kTileDim, (N + kTileDim - 1) / kTileDim);
        matrixMultiplyKernel_V1<<<grid, block>>>(
            d_a, d_b, d_c, M, N, K, 1.0, 0.0);
      },
      [&] {
        CUDA_CHECK(hipMemcpy(h_c, d_c, C_SIZE, hipMemcpyDeviceToHost));
        dbg(utils::checkEqual(h_c, real_c, M * N));
      });

  utils::performance<GpuTimer>(
      "matrixMultiplyKernel_V1_32x32x32",
      repeats,
      [&] { utils::fill_n(d_c, M * N, 0.0); },
      [&] {
        constexpr int kTileDim = 32;
        dim3 block(kTileDim, kTileDim);
        dim3 grid((M + kTileDim - 1) / kTileDim, (N + kTileDim - 1) / kTileDim);
        matrixMultiplyKernel_V1<<<grid, block>>>(
            d_a, d_b, d_c, M, N, K, 1.0, 0.0);
      },
      [&] {
        CUDA_CHECK(hipMemcpy(h_c, d_c, C_SIZE, hipMemcpyDeviceToHost));
        dbg(utils::checkEqual(h_c, real_c, M * N));
      });

  utils::performance<GpuTimer>(
      "matrixMultiplyKernel_V2_16x16x16",
      repeats,
      [&] { utils::fill_n(d_c, M * N, 0.0); },
      [&] {
        constexpr int kTileDim = 16;
        dim3 block(kTileDim, kTileDim);
        dim3 grid((M + kTileDim - 1) / kTileDim, (N + kTileDim - 1) / kTileDim);
        matrixMultiplyKernel_V2<kTileDim>
            <<<grid, block>>>(d_a, d_b, d_c, M, N, K, 1.0, 0.0);
      },
      [&] {
        CUDA_CHECK(hipMemcpy(h_c, d_c, C_SIZE, hipMemcpyDeviceToHost));
        dbg(utils::checkEqual(h_c, real_c, M * N));
      });

  utils::performance<GpuTimer>(
      "matrixMultiplyKernel_V2_32x32x32",
      repeats,
      [&] { utils::fill_n(d_c, M * N, 0.0); },
      [&] {
        constexpr int kTileDim = 32;
        dim3 block(kTileDim, kTileDim);
        dim3 grid((M + kTileDim - 1) / kTileDim, (N + kTileDim - 1) / kTileDim);
        matrixMultiplyKernel_V2<kTileDim>
            <<<grid, block>>>(d_a, d_b, d_c, M, N, K, 1.0, 0.0);
      },
      [&] {
        CUDA_CHECK(hipMemcpy(h_c, d_c, C_SIZE, hipMemcpyDeviceToHost));
        dbg(utils::checkEqual(h_c, real_c, M * N));
      });

  // utils::performance<GpuTimer>(
  //     "matrixMultiplyKernel_V3_16x16x64",
  //     repeats,
  //     [&] { utils::fill_n(d_c, M * N, 0.0); },
  //     [&] {
  //       constexpr int kTileDim = 16;
  //       dim3 block(kTileDim, kTileDim);
  //       dim3 grid((M + kTileDim - 1) / kTileDim / 4,
  //                 (N + kTileDim - 1) / kTileDim);
  //       matrixMultiplyKernel_V3<kTileDim>
  //           <<<grid, block>>>(d_a, d_b, d_c, M, N, K, 1.0, 0.0);
  //     },
  //     [&] {
  //       CUDA_CHECK(hipMemcpy(h_c, d_c, C_SIZE, hipMemcpyDeviceToHost));
  //       dbg(utils::checkEqual(h_c, real_c, M * N));
  //     });
}
