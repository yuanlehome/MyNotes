#include "hip/hip_runtime.h"
#include "all.h"
#include "common.h"
#include "kernel_utils.cu.h"

constexpr int kTileDim = 32;

// Naive solution as baseline
// block(kTileDim, kTileDim)
// grid(M / kTileDim, N / kTileDim)
__global__ void matrixMultiplyKernel_V1(
    const float* A, const float* B, float* C, int M, int N, int K) {
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  if (col < N && row < M) {
    float sum = 0.0;
    for (int k = 0; k < K; k++) {
      sum += A[k + row * K] * B[col + k * N];
    }
    C[col + row * N] = sum;
  }
}

// Use shared memory
// block(kTileDim, kTileDim)
// grid(M / kTileDim, N / kTileDim)
__global__ void matrixMultiplyKernel_V2(
    const float* A, const float* B, float* C, int M, int N, int K) {
  __shared__ float s_a[kTileDim][kTileDim];
  __shared__ float s_b[kTileDim][kTileDim + 1];
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  if (col < N && row < M) {
    float sum = 0.0;
    for (int k = 0; k < K; k += kTileDim) {
      s_a[threadIdx.y][threadIdx.x] = A[k + threadIdx.x + row * K];
      s_b[threadIdx.y][threadIdx.x] = B[col + (k + threadIdx.y) * N];
      __syncthreads();

      for (int i = 0; i < kTileDim; i++) {
        sum += s_a[threadIdx.y][i] * s_b[i][threadIdx.x];
      }
      __syncthreads();
    }
    C[col + row * N] = sum;
  }
}

void gemm_naive() {
  constexpr uint32_t M = 1024;
  constexpr uint32_t N = 1024;
  constexpr uint32_t K = 512;

  constexpr uint32_t A_SIZE = sizeof(DATA_TYPE) * M * K;
  constexpr uint32_t B_SIZE = sizeof(DATA_TYPE) * K * N;
  constexpr uint32_t C_SIZE = sizeof(DATA_TYPE) * M * N;

  CPUMallocWrapper cpu_allocator;
  DATA_TYPE* h_a = (DATA_TYPE*)cpu_allocator.allocate(A_SIZE);
  DATA_TYPE* h_b = (DATA_TYPE*)cpu_allocator.allocate(B_SIZE);
  DATA_TYPE* h_c = (DATA_TYPE*)cpu_allocator.allocate(C_SIZE);
  std::fill_n(h_a, M * K, 0.5);
  std::fill_n(h_b, K * N, 0.3);
  std::fill_n(h_c, M * N, 0.0);

  GPUMallocWrapper gpu_allocator;
  DATA_TYPE* d_a = (DATA_TYPE*)gpu_allocator.allocate(A_SIZE);
  DATA_TYPE* d_b = (DATA_TYPE*)gpu_allocator.allocate(B_SIZE);
  DATA_TYPE* d_c = (DATA_TYPE*)gpu_allocator.allocate(C_SIZE);
  CUDA_CHECK(hipMemcpy(d_a, h_a, A_SIZE, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_b, h_b, B_SIZE, hipMemcpyHostToDevice));
  utils::fill_n(d_c, M * N, 0.0);

  // CPU results
  DATA_TYPE* real_c = (DATA_TYPE*)cpu_allocator.allocate(C_SIZE);
  std::fill_n(real_c, M * N, 0.0);
  utils::matrixMultiply(h_a, h_b, real_c, M, N, K);

  // GPU results
  dim3 block(kTileDim, kTileDim);
  dim3 grid((M + kTileDim - 1) / kTileDim, (N + kTileDim - 1) / kTileDim);

  utils::performance<GPUTimer>(
      "matrixMultiplyKernel_V1",
      repeats,
      [&] { utils::fill_n(d_c, M * N, 0.0); },
      [&] { matrixMultiplyKernel_V1<<<grid, block>>>(d_a, d_b, d_c, M, N, K); },
      [&] {
        CUDA_CHECK(hipMemcpy(h_c, d_c, C_SIZE, hipMemcpyDeviceToHost));
        dbg(utils::checkEqual(h_c, real_c, M * N));
      });

  utils::performance<GPUTimer>(
      "matrixMultiplyKernel_V2",
      repeats,
      [&] { utils::fill_n(d_c, M * N, 0.0); },
      [&] { matrixMultiplyKernel_V2<<<grid, block>>>(d_a, d_b, d_c, M, N, K); },
      [&] {
        CUDA_CHECK(hipMemcpy(h_c, d_c, C_SIZE, hipMemcpyDeviceToHost));
        dbg(utils::checkEqual(h_c, real_c, M * N));
      });
}
