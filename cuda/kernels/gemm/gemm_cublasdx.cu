#include "hip/hip_runtime.h"
// https://docs.nvidia.com/cuda/cublasdx/index.html

#include <iostream>
#include <random>
#include <type_traits>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <cublasdx.hpp>

#define CUDA_CHECK_AND_EXIT(error)                                      \
  {                                                                     \
    auto status = static_cast<hipError_t>(error);                      \
    if (status != hipSuccess) {                                        \
      std::cout << hipGetErrorString(status) << " " << __FILE__ << ":" \
                << __LINE__ << std::endl;                               \
      std::exit(status);                                                \
    }                                                                   \
  }

template <typename T>
std::vector<T> get_random_data(const float min,
                               const float max,
                               const size_t size) {
  std::default_random_engine e;
  std::uniform_real_distribution<float> u(min, max);

  std::vector<T> ret(size);
  for (auto& v : ret) {
    v = static_cast<T>(u(e));
  }
  return ret;
}

namespace example {
template <class T>
inline __device__ void naive_copy(T* dest, const T* src, unsigned int size) {
  if ((threadIdx.x == 0) && (threadIdx.y == 0) && (threadIdx.z == 0)) {
    // Note: This copies values in padding too
    for (unsigned int idx = 0; idx < size; ++idx) {
      dest[idx] = src[idx];
    }
  }
}
template <class BLAS, class ValueType = typename BLAS::value_type>
struct io {
  using value_type = ValueType;

  template <class T>
  static inline __device__ void load(T* shared_output,
                                     const value_type* global_input,
                                     const unsigned int size) {
    naive_copy(
        reinterpret_cast<value_type*>(shared_output), global_input, size);
  }

  template <class T>
  static inline __device__ void store(value_type* global_output,
                                      const T* shared_input,
                                      const unsigned int size) {
    naive_copy(
        global_output, reinterpret_cast<const value_type*>(shared_input), size);
  }
};
}  // namespace example

template <class BLAS1,
          class BLAS2,
          class ValueType = typename BLAS1::value_type>
__launch_bounds__(BLAS1::max_threads_per_block)  //
    __global__                                   //
    void gemm_kernel(const ValueType alpha1,
                     const ValueType* a,
                     const ValueType* b,
                     const ValueType beta1,
                     const ValueType* c,
                     const ValueType alpha2,
                     const ValueType* d,
                     const ValueType beta2,
                     const ValueType* f,
                     ValueType* output) {
  using value_type = ValueType;
  extern __shared__ __align__(16) char smem[];

  // Matrix C is the first in shared memory, because it's reused in the 2nd
  // GEMM. Moreover, matrices A and B might have different sizes than F and D.
  value_type* smem_c = reinterpret_cast<value_type*>(smem);
  value_type* smem_a = reinterpret_cast<value_type*>(smem) + BLAS1::c_size;
  value_type* smem_b =
      reinterpret_cast<value_type*>(smem) + BLAS1::c_size + BLAS1::a_size;

  example::io<BLAS1>::load(smem_a, a, BLAS1::a_size);
  example::io<BLAS1>::load(smem_b, b, BLAS1::b_size);
  example::io<BLAS1>::load(smem_c, c, BLAS1::c_size);
  __syncthreads();

  BLAS1().execute(alpha1, smem_a, smem_b, beta1, smem_c);
  __syncthreads();

  static_assert((BLAS1::c_size == BLAS2::a_size),
                "The sizes of C matrix are different in BLAS1 and BLAS2");
  value_type* smem_d = smem_c + BLAS2::a_size;
  value_type* smem_f = smem_c + BLAS2::a_size + BLAS2::b_size;
  example::io<BLAS2>::load(smem_d, d, BLAS2::b_size);
  example::io<BLAS2>::load(smem_f, f, BLAS2::c_size);
  __syncthreads();

  BLAS2().execute(alpha2, smem_c, smem_d, beta2, smem_f);

  __syncthreads();
  example::io<BLAS2>::store(output, smem_f, BLAS2::c_size);
}

//             1) C = alpha1 * (A * B) + beta1 * C
//             2) F = alpha2 * (C * D) + beta2 * F
int simple_gemm() {
  // Parameters m1, n1, k1 define the dimensions of matrices A, B, and C
  constexpr unsigned int m1 = 64;
  constexpr unsigned int n1 = 64;
  constexpr unsigned int k1 = 64;

  // Parameters m2, n2, k2 define the dimensions of matrices C, D and F
  // Note: (m1, n1) and (m2, k2) must be equal as describe the same matrix
  // (matrix C)
  constexpr unsigned int m2 = m1;
  constexpr unsigned int n2 = 128;
  constexpr unsigned int k2 = n1;

  // The logical dimensions of matrix A are: [m1, k1] (m rows, k columns)
  // The logical dimensions of matrix B are: [k1, n1]
  // The logical dimensions of matrix C are: [m1, n1]
  constexpr auto a_transpose_mode = cublasdx::transpose_mode::non_transposed;
  constexpr auto b_transpose_mode = cublasdx::transpose_mode::non_transposed;

  // The logical dimensions of matrix C are: [m2, k2] == [m1, n1]
  // The logical dimensions of matrix D are: [k2, n2]
  // The logical dimensions of matrix F are: [m2, n2]
  constexpr auto c_transpose_mode = cublasdx::transpose_mode::non_transposed;
  constexpr auto d_transpose_mode = cublasdx::transpose_mode::non_transposed;

  // Use the same block size for both GEMM operations, so BLAS1::block_dim ==
  // BLAS2::block_dim which simplifies the example.
  constexpr unsigned int block_size = 128;

  using BLAS1 =
      decltype(cublasdx::Size<m1, n1, k1>() + cublasdx::Precision<__half>() +
               cublasdx::Type<cublasdx::type::real>() +
               cublasdx::Function<cublasdx::function::MM>() +
               cublasdx::TransposeMode<a_transpose_mode, b_transpose_mode>() +
               cublasdx::Block() + cublasdx::BlockDim<block_size>() +
               cublasdx::SM<800>());
  using BLAS2 =
      decltype(cublasdx::Size<m2, n2, k2>() + cublasdx::Precision<__half>() +
               cublasdx::Type<cublasdx::type::real>() +
               cublasdx::Function<cublasdx::function::MM>() +
               cublasdx::TransposeMode<c_transpose_mode, d_transpose_mode>() +
               cublasdx::Block() + cublasdx::BlockDim<block_size>() +
               cublasdx::SM<800>());
  using value_type = typename BLAS1::value_type;

  // alpha and beta for the first GEMM
  value_type alpha1 = 1.0;
  value_type beta1 = 0.0;

  // alpha and beta for the 2nd GEMM
  value_type alpha2 = 1.0;
  value_type beta2 = 1.0;

  // Allocate managed memory for a, b, c, d, f and output
  value_type* inputs;
  value_type* output;
  auto inputs_size = BLAS1::a_size + BLAS1::b_size + BLAS1::c_size +
                     BLAS2::b_size + BLAS2::c_size;
  auto inputs_size_bytes = inputs_size * sizeof(value_type);
  CUDA_CHECK_AND_EXIT(hipMalloc(&inputs, inputs_size_bytes));
  CUDA_CHECK_AND_EXIT(hipMalloc(&output, BLAS2::c_size * sizeof(value_type)));

  value_type* a = inputs;
  value_type* b = a + (BLAS1::a_size);
  value_type* c =
      b + (BLAS1::b_size);  // C matrix for BLAS1, A matrix for BLAS2
  value_type* d = c + (BLAS1::c_size);  // D is B matrix for BLAS2
  value_type* f = d + (BLAS2::b_size);  // F is C matrix for BLAS2

  // Fill the A, B, C matrices with random values
  auto host_a = get_random_data<value_type>(0.1, 1.0, BLAS1::a_size);
  auto host_b = get_random_data<value_type>(0.1, 1.0, BLAS1::b_size);
  auto host_c = get_random_data<value_type>(0.1, 1.0, BLAS1::c_size);
  auto host_d = get_random_data<value_type>(1.0, 2.0, BLAS2::b_size);
  auto host_f = get_random_data<value_type>(1.0, 10.0, BLAS2::c_size);
  CUDA_CHECK_AND_EXIT(hipMemcpy(a,
                                 host_a.data(),
                                 BLAS1::a_size * sizeof(value_type),
                                 hipMemcpyHostToDevice));
  CUDA_CHECK_AND_EXIT(hipMemcpy(b,
                                 host_b.data(),
                                 BLAS1::b_size * sizeof(value_type),
                                 hipMemcpyHostToDevice));
  CUDA_CHECK_AND_EXIT(hipMemcpy(c,
                                 host_c.data(),
                                 BLAS1::c_size * sizeof(value_type),
                                 hipMemcpyHostToDevice));
  CUDA_CHECK_AND_EXIT(hipMemcpy(d,
                                 host_d.data(),
                                 BLAS2::b_size * sizeof(value_type),
                                 hipMemcpyHostToDevice));
  CUDA_CHECK_AND_EXIT(hipMemcpy(f,
                                 host_f.data(),
                                 BLAS2::c_size * sizeof(value_type),
                                 hipMemcpyHostToDevice));
  CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

  // Increase max dynamic shared memory for the kernel if needed
  const auto shared_memory =
      std::max<size_t>(BLAS1::shared_memory_size, BLAS2::shared_memory_size);
  CUDA_CHECK_AND_EXIT(
      hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_kernel<BLAS1), BLAS2>,
                           hipFuncAttributeMaxDynamicSharedMemorySize,
                           shared_memory));

  // Execute kernel
  gemm_kernel<BLAS1, BLAS2><<<1, BLAS1::block_dim, shared_memory>>>(
      alpha1, a, b, beta1, c, alpha2, d, beta2, f, output);
  CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

  // Copy results back to host
  std::vector<value_type> host_output(BLAS2::c_size);
  CUDA_CHECK_AND_EXIT(hipMemcpy(host_output.data(),
                                 output,
                                 BLAS2::c_size * sizeof(value_type),
                                 hipMemcpyDeviceToHost));
  CUDA_CHECK_AND_EXIT(hipDeviceSynchronize());

  // Free device memory
  CUDA_CHECK_AND_EXIT(hipFree(inputs));
  CUDA_CHECK_AND_EXIT(hipFree(output));
  return 1;
}

int main(int, char**) {
  simple_gemm();
  printf("run success!\n");
}
