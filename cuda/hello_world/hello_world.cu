#include "hip/hip_runtime.h"
#include <stdio.h>

#include "kernel_caller_declare.h"

__global__ void hello_world_v1() {
  const int bid = blockIdx.x;
  const int tid = threadIdx.x;
  printf("Hello world from gpu int block %d and thread %d.\n", bid, tid);
}

__global__ void hello_world_v2() {
  const int bid_x = blockIdx.x;
  const int bid_y = blockIdx.y;
  const int tid_x = threadIdx.x;
  const int tid_y = threadIdx.y;
  printf("Hello world from gpu int block (%d, %d) and thread (%d, %d).\n",
         bid_x,
         bid_y,
         tid_x,
         tid_y);
}

void print_hello_world() {
  // hello_world_v1<<<2, 4>>>();

  dim3 block_size(2, 4);
  dim3 grid_size(1, 1);
  hello_world_v2<<<grid_size, block_size>>>();
  hipDeviceSynchronize();
}
