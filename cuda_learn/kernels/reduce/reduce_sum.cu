#include "hip/hip_runtime.h"
#include "dbg.h"

#include "common.h"
#include "kernel_caller_declare.h"
#include "kernel_utils.cuh"

constexpr DATA_TYPE a = 1.23;

// 数值错误
// Add x[start...end] 左闭右闭
DATA_TYPE reduceSumOnCPU_V1(const DATA_TYPE* x,
                            const int start,
                            const int end) {
  DATA_TYPE sum{0.0};
  for (int i = start; i <= end; i++) {
    // 大数加小数
    sum += x[i];
  }
  return sum;
}

// 数值正确 不修改原数组
// Add x[start...end] 左闭右闭
DATA_TYPE reduceSumOnCPU_V2(const DATA_TYPE* x,
                            const int start,
                            const int end) {
  if (start > end) return DATA_TYPE{};
  if (start == end) return x[start];
  const int p = (start + end) / 2;
  // 递归
  return reduceSumOnCPU_V2(x, start, p) + reduceSumOnCPU_V2(x, p + 1, end);
}

// 数值正确 修改原数组
// Add x[start...end] 左闭右闭
DATA_TYPE reduceSumOnCPU_V3(DATA_TYPE* x, int start, int end) {
  while (start < end) {
    // 双指针
    int i = start, j = end;
    while (i < j) {
      x[i++] += x[j--];
    }
    end = i == j ? i : i - 1;
  }
  return x[start];
}

// 数值正确 需二次 reduce 要求数据个数为 BLOCK_SIZE 的整数倍 改变原数组
// 每个 block 负责一块内存数据的 reduce
__global__ void reduceSumOnGPU_V1(DATA_TYPE* d_x, DATA_TYPE* d_y) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  DATA_TYPE* x = d_x + blockDim.x * bid;

  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      x[tid] += x[tid + offset];
    }
    __syncthreads();
  }

  if (tid == 0) {
    d_y[bid] = x[0];
  }
}

// 数值正确 需二次 reduce 不要求数据个数为 BLOCK_SIZE 的整数倍 不改变原数组
// 每个 block 负责一块内存数据的 reduce 使用共享内存
__global__ void reduceSumOnGPU_V2(const DATA_TYPE* d_x,
                                  DATA_TYPE* d_y,
                                  const int N) {
  extern __shared__ DATA_TYPE s_y[];
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const int idx = tid + blockDim.x * bid;
  s_y[tid] = idx < N ? d_x[idx] : 0.0;
  __syncthreads();

  for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
    if (tid < offset) {
      s_y[tid] += s_y[tid + offset];
    }
    __syncthreads();
  }

  if (tid == 0) {
    d_y[bid] = s_y[0];
  }
}

// 数值正确 需二次 reduce 要求数据个数为 BLOCK_SIZE 的整数倍 不改变原数组
// 每个 block 负责一块内存数据的 reduce
__global__ void reduceSumOnGPU_V3(const DATA_TYPE* d_x, DATA_TYPE* d_y) {
  const int tid = threadIdx.x;
  const int bid = blockIdx.x;
  const DATA_TYPE* x = d_x + blockDim.x * bid;

  DATA_TYPE value = blockReduceSum(x[tid]);

  if (tid == 0) {
    d_y[bid] = value;
  }
}

void reduceSum() {
  constexpr uint32_t N = 1e8;
  constexpr uint32_t M = sizeof(DATA_TYPE) * N;

  MallocWrapper cpu_allocator;
  DATA_TYPE* h_x = (DATA_TYPE*)cpu_allocator.allocate(M);
  std::fill_n(h_x, N, a);

  // Timer cpu_timer;
  // float total_time = 0.0;
  // DATA_TYPE sum_on_cpu;
  // for (size_t i = 0; i < repeats; i++) {
  //   cpu_timer.start();
  //   sum_on_cpu = reduceSumOnCPU_V1(h_x, 0, N - 1);
  //   cpu_timer.stop();
  //   total_time += cpu_timer.elapsedTime();
  // }
  // dbg(sum_on_cpu);
  // std::printf("reduceSumOnCPU_V1 cost time: %f ms\n", total_time / repeats);

  // total_time = 0.0;
  // for (size_t i = 0; i < repeats; i++) {
  //   cpu_timer.start();
  //   sum_on_cpu = reduceSumOnCPU_V2(h_x, 0, N - 1);
  //   cpu_timer.stop();
  //   total_time += cpu_timer.elapsedTime();
  // }
  // dbg(sum_on_cpu);
  // std::printf("reduceSumOnCPU_V2 cost time: %f ms\n", total_time / repeats);

  // total_time = 0.0;
  // for (size_t i = 0; i < repeats; i++) {
  //   cpu_timer.start();
  //   sum_on_cpu = reduceSumOnCPU_V3(h_x, 0, N - 1);
  //   cpu_timer.stop();
  //   total_time += cpu_timer.elapsedTime();
  //   std::fill_n(h_x, N, a);  // 恢复原数组
  // }
  // dbg(sum_on_cpu);
  // std::printf("reduceSumOnCPU_V3 cost time: %f ms\n", total_time / repeats);

  const uint32_t block_size = BLOCK_SIZE;
  const uint32_t grid_size = (N + block_size - 1) / block_size;
  dbg(block_size, grid_size);
  dim3 block(block_size);
  dim3 grid(grid_size);

  GPUMallocWrapper gpu_allocator;
  DATA_TYPE* d_x = (DATA_TYPE*)gpu_allocator.allocate(M);
  CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));

  constexpr uint32_t RES_SIZE = sizeof(DATA_TYPE) * grid_size;
  DATA_TYPE* d_y = (DATA_TYPE*)gpu_allocator.allocate(RES_SIZE);
  DATA_TYPE* h_y = (DATA_TYPE*)cpu_allocator.allocate(RES_SIZE);

  GPUTimer gpu_timer;
  float total_time = 0.0;
  for (size_t i = 0; i < repeats; i++) {
    gpu_timer.start();
    reduceSumOnGPU_V1<<<grid, block>>>(d_x, d_y);
    gpu_timer.stop();
    total_time += gpu_timer.elapsedTime();
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));  // 恢复原数组
  }
  dbg(total_time, gpu_timer.totalTime());
  CHECK(hipMemcpy(h_y, d_y, RES_SIZE, hipMemcpyDeviceToHost));
  dbg(*h_y);
  std::printf("reduceSumOnGPU_V1 cost time: %f ms\n", total_time / repeats);

  total_time = 0.0;
  for (size_t i = 0; i < repeats; i++) {
    gpu_timer.start();
    reduceSumOnGPU_V2<<<grid, block, sizeof(DATA_TYPE) * block_size>>>(
        d_x, d_y, N);
    gpu_timer.stop();
    total_time += gpu_timer.elapsedTime();
  }
  dbg(total_time, gpu_timer.totalTime());
  CHECK(hipMemcpy(h_y, d_y, RES_SIZE, hipMemcpyDeviceToHost));
  dbg(*(h_y + 12354));
  std::printf("reduceSumOnGPU_V2 cost time: %f ms\n", total_time / repeats);

  total_time = 0.0;
  for (size_t i = 0; i < repeats; i++) {
    gpu_timer.start();
    reduceSumOnGPU_V3<<<grid, block>>>(d_x, d_y);
    gpu_timer.stop();
    total_time += gpu_timer.elapsedTime();
  }
  dbg(total_time, gpu_timer.totalTime());
  CHECK(hipMemcpy(h_y, d_y, RES_SIZE, hipMemcpyDeviceToHost));
  dbg(*(h_y + 12354));
  std::printf("reduceSumOnGPU_V3 cost time: %f ms\n", total_time / repeats);
}
