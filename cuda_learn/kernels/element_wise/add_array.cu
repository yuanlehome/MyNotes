#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdint>

#include "dbg.h"

#include "common.h"
#include "kernel_caller_declare.h"
#include "kernel_utils.cuh"

constexpr DATA_TYPE a = 1.23;
constexpr DATA_TYPE b = 2.34;
constexpr DATA_TYPE c = 3.57;

void addArrayOnCPU(const DATA_TYPE* x,
                   const DATA_TYPE* y,
                   DATA_TYPE* z,
                   const uint32_t N) {
  for (size_t i = 0; i < N; i++) {
    z[i] = x[i] + y[i];
  }
}

__device__ void add(const DATA_TYPE a, const DATA_TYPE b, DATA_TYPE* c) {
  *c = a + b;
}

__global__ void addArrayOnGPU(const DATA_TYPE* x,
                              const DATA_TYPE* y,
                              DATA_TYPE* z,
                              const uint32_t N) {
  const uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < N) {
    add(x[tid], y[tid], &z[tid]);
  }
}

void addArray() {
  constexpr uint32_t N = 1e8 + 1;
  constexpr uint32_t M = sizeof(DATA_TYPE) * N;

  MallocWrapper cpu_allocator;
  DATA_TYPE* h_x = (DATA_TYPE*)cpu_allocator.allocate(M);
  DATA_TYPE* h_y = (DATA_TYPE*)cpu_allocator.allocate(M);
  DATA_TYPE* h_z = (DATA_TYPE*)cpu_allocator.allocate(M);

  std::fill_n(h_x, N, a);
  std::fill_n(h_y, N, b);

  for (size_t i = 0; i < warm_up; i++) {
    addArrayOnCPU(h_x, h_y, h_z, N);
  }
  Timer cpu_timer;
  float total_time = 0.0;
  for (size_t i = 0; i < repeats; i++) {
    cpu_timer.start();
    addArrayOnCPU(h_x, h_y, h_z, N);
    cpu_timer.stop();
    total_time += cpu_timer.elapsedTime();
  }
  dbg(total_time, cpu_timer.totalTime());
  std::printf("addArrayOnCPU cost time: %f ms\n", total_time / repeats);
  dbg(checkEqual(h_z, N, c));

  GPUMallocWrapper gpu_allocator;
  DATA_TYPE* d_x = (DATA_TYPE*)gpu_allocator.allocate(M);
  DATA_TYPE* d_y = (DATA_TYPE*)gpu_allocator.allocate(M);
  DATA_TYPE* d_z = (DATA_TYPE*)gpu_allocator.allocate(M);

  CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice));

  const uint32_t block_size = 128;
  const uint32_t grid_size = (N + block_size - 1) / block_size;
  dbg(block_size, grid_size);
  dim3 block(block_size);
  dim3 grid(grid_size);

  for (size_t i = 0; i < warm_up; i++) {
    addArrayOnGPU<<<grid, block>>>(d_x, d_y, d_z, N);
  }
  GPUTimer gpu_timer;
  total_time = 0.0;
  for (size_t i = 0; i < repeats; i++) {
    gpu_timer.start();
    addArrayOnGPU<<<grid, block>>>(d_x, d_y, d_z, N);
    gpu_timer.stop();
    total_time += gpu_timer.elapsedTime();
  }
  std::printf("addArrayOnGPU cost time: %f ms\n", total_time / repeats);

  CHECK(hipMemcpy(h_z, d_z, M, hipMemcpyDeviceToHost));
  dbg(checkEqual(h_z, N, c));
}
